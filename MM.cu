
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//#include <cblas.h>//cuda用線形計算ライブラリ 
//static const int M_SIZE = 3;//matrix size
//static const int B_SIZE = 1024;//block size
#define M_SIZE 10
#define B_SIZE 1024
//CPU プロトタイプ
void matrixMul(int *HM1, int *HM2, int *HM3);
void matrixZeros(int *HM);
void matrixTranspose(int *iMat, int*oMat);
int matrixDiffCount(int *HM1, int *HM2);
void printHM(int *HM);
//CUDA プロトタイプ
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3);
__global__ void cudaMatrixMulShared(int *GM1, int *GM2, int *GM3);
//メイン関数
int main(void){
    srand((unsigned)time(NULL));
    //Host Memory に Matrix 確保 HM1 = HM2 * HM3
    int *HM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *HM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *HM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    //Host Memoryにデータ格納
    matrixZeros(HM1);
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        HM2[i] = rand()%256;
        HM3[i] = rand()%256;
    }
    //Global Memory 確保
    int *GM1,*GM2,*GM3;
    hipMalloc((void **)&GM1, sizeof(int) * M_SIZE * M_SIZE);
    hipMalloc((void **)&GM2, sizeof(int) * M_SIZE * M_SIZE);
    hipMalloc((void **)&GM3, sizeof(int) * M_SIZE * M_SIZE);
    //GlobalMemoryにデータ格納
    hipMemcpy(GM1, HM1, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(GM2, HM2, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(GM3, HM3, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    //CPUでの計算
    hipEvent_t cpuStartTime,cpuStopTime;
    float cpuTime;
    hipEventCreate(&cpuStartTime);
    hipEventCreate(&cpuStopTime);
    hipEventRecord(cpuStartTime, 0);
    matrixMul(HM1, HM2, HM3);
    hipEventRecord(cpuStopTime, 0);
    hipEventSynchronize(cpuStopTime);
    hipEventElapsedTime(&cpuTime, cpuStartTime, cpuStopTime);
    //CUDAでの計算
    hipEvent_t cudaStartTime, cudaStopTime;
    float cudaTime;
    int DgSize = M_SIZE*M_SIZE/B_SIZE;
    if(DgSize == 0){
        DgSize = 1;
    }
    dim3 Dg(1, 1, 1), Db(100, 1, 1);
    hipEventCreate(&cudaStartTime);
    hipEventCreate(&cudaStopTime);
    hipEventRecord(cudaStartTime, 0);
    cudaMatrixMul <<< Dg, Db>>> (GM1, GM2, GM3);
    hipEventRecord(cudaStopTime, 0);
    hipEventSynchronize(cudaStopTime);
    hipEventElapsedTime(&cudaTime, cudaStartTime, cudaStopTime);
    //Cuda計算結果をHostMemoryにコピー
    int *cudaHM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *cudaHM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *cudaHM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    hipMemcpy(cudaHM1, GM1, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(cudaHM2, GM2, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(cudaHM3, GM3, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    //標準出力
    printf("M_SIZE:%d\n",M_SIZE);
    if(matrixDiffCount(HM1,cudaHM1)){
        puts("CPUとGPUの計算結果は一致しました");
    }else{
        puts("CPUとGPUの計算結果は一致しませんでした");
    }
    if(M_SIZE <= 10){
        puts("M1");
        printHM(HM1);
        puts("M2");
        printHM(HM2);
        puts("M3");
        printHM(HM3);
        puts("cudaM");
        printHM(cudaHM1);
        puts("cuda2");
        printHM(cudaHM2);
        puts("cuda3");
        printHM(cudaHM3);
    }
    printf("CPU:Time = %f\n", cpuTime);
    printf("GPU:Time = %f\n", cudaTime);
    //Host Memory開放
    free(HM1);
    free(HM2);
    free(HM3);
    //Global Memory開放
    hipFree(GM1);
    hipFree(GM2);
    hipFree(GM3);
    return 0;
}
//CUDA版行列の積
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int row = id/M_SIZE;
    int column = id%M_SIZE;
    int x=0;
    for(int i=0; i<M_SIZE; i++){
        x += GM2[row*M_SIZE+i] * GM3[i*M_SIZE+column];
    }
    GM1[id] = x;
}
//CUDA,SharedMemory使用版行列の積
__global__ void cudaMatrixMulShared(int *GM1, int *GM2, int *GM3){
    __shared__ int SM2[1024], SM3[1024];
    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    SM2[tid] = GM2[id];
    SM3[tid] = GM3[id];
    __syncthreads();
    int row = id/M_SIZE;
    int column = id%M_SIZE;
    int x=0;
    for(int i=0; i<M_SIZE; i++){
        x += SM2[row*M_SIZE+i] * SM3[i*M_SIZE+column];
    }
    GM1[id] = x;
}
//CPU版行列の積
void matrixMul(int *HM1, int *HM2, int *HM3){
    int tmpHM[M_SIZE*M_SIZE];
    matrixZeros(tmpHM);
    matrixTranspose(HM3,tmpHM);//転置
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE; j++){
            for(int k=0; k<M_SIZE; k++){
                HM1[i*M_SIZE + j] += HM2[i*M_SIZE + k] * tmpHM[j*M_SIZE + k];
            }
        }
    }
}
void matrixZeros(int *HM){
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        HM[i]=0;
    }
}
//転置行列
void matrixTranspose(int *iMat, int *oMat){
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE; j++){
            oMat[j*M_SIZE + i] = iMat[i*M_SIZE + j];
        }
    }
}
int matrixDiffCount(int *HM1, int *HM2){
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        if(HM1[i] - HM2[i]){
            return 0;
        }
    }
    return 1;
}
void printHM(int *HM){
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE;j++){
            printf("%d,",HM[i*M_SIZE + j]);
        }
        printf("\n");
    }
    printf("\n");
}
