
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//#include <cblas.h>//cuda用線形計算ライブラリ 
//static const int M_SIZE = 3;//matrix size
//static const int B_SIZE = 1024;//block size
#define M_SIZE 1024
#define B_SIZE 1024
#define SUB_SIZE 6
//CPU プロトタイプ
void matrixMul(int *HM1, int *HM2, int *HM3);
void matrixZeros(int *HM);
void matrixTranspose(int *iMat, int*oMat);
int matrixDiffCount(int *HM1, int *HM2);
void printHM(int *HM);
//CUDA プロトタイプ
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3);
__global__ void cudaMatrixMulShared(int *GM1, int *GM2, int *GM3);
//メイン関数
int main(void){
    srand((unsigned)time(NULL));
    //Host Memory に Matrix 確保 HM1 = HM2 * HM3
    int *HM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *HM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *HM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    //Host Memoryにデータ格納
    matrixZeros(HM1);
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        HM2[i] = rand()%256;
        HM3[i] = rand()%256;
    }
    //Global Memory 確保
    int *GM1,*GM2,*GM3;
    hipMalloc((void **)&GM1, sizeof(int) * M_SIZE * M_SIZE);
    hipMalloc((void **)&GM2, sizeof(int) * M_SIZE * M_SIZE);
    hipMalloc((void **)&GM3, sizeof(int) * M_SIZE * M_SIZE);
    //GlobalMemoryにデータ格納
    hipMemcpy(GM1, HM1, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(GM2, HM2, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    int *TM = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    matrixTranspose(HM3,TM);
    hipMemcpy(GM3, TM, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
    //CPUでの計算
    hipEvent_t cpuStartTime,cpuStopTime;
    float cpuTime;
    hipEventCreate(&cpuStartTime);
    hipEventCreate(&cpuStopTime);
    hipEventRecord(cpuStartTime, 0);
    matrixMul(HM1, HM2, HM3);
    hipEventRecord(cpuStopTime, 0);
    hipEventSynchronize(cpuStopTime);
    hipEventElapsedTime(&cpuTime, cpuStartTime, cpuStopTime);
    //CUDAでの計算
    hipEvent_t cudaStartTime, cudaStopTime;
    float cudaTime;
    dim3 Dg(M_SIZE/SUB_SIZE, M_SIZE/SUB_SIZE, 1), Db(1024, 1, 1);
    hipEventCreate(&cudaStartTime);
    hipEventCreate(&cudaStopTime);
    hipEventRecord(cudaStartTime, 0);
    cudaMatrixMulShared <<< Dg, Db, 40000>>> (GM1, GM2, GM3);
    hipEventRecord(cudaStopTime, 0);
    hipEventSynchronize(cudaStopTime);
    hipEventElapsedTime(&cudaTime, cudaStartTime, cudaStopTime);
    //Cuda計算結果をHostMemoryにコピー
    int *cudaHM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *cudaHM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    int *cudaHM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
    hipMemcpy(cudaHM1, GM1, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(cudaHM2, GM2, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(cudaHM3, GM3, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyDeviceToHost);
    //標準出力
    printf("M_SIZE:%d\n",M_SIZE);
    if(matrixDiffCount(HM1,cudaHM1)){
        puts("CPUとGPUの計算結果は一致しました");
    }else{
        puts("CPUとGPUの計算結果は一致しませんでした");
    }
    if(M_SIZE <= 10){
        puts("M1");
        printHM(HM1);
        puts("M2");
        printHM(HM2);
        puts("M3");
        printHM(HM3);
        puts("cudaM");
        printHM(cudaHM1);
        puts("cuda2");
        printHM(cudaHM2);
        puts("cuda3");
        printHM(cudaHM3);
    }
    printf("CPU:Time = %f\n", cpuTime);
    printf("GPU:Time = %f\n", cudaTime);
    //Host Memory開放
    free(HM1);
    free(HM2);
    free(HM3);
    //Global Memory開放
    hipFree(GM1);
    hipFree(GM2);
    hipFree(GM3);
    return 0;
}
//CUDA版行列の積
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int row = id/M_SIZE;
    int column = id%M_SIZE;
    int x=0;
    for(int i=0; i<M_SIZE; i++){
        x += GM2[row*M_SIZE+i] * GM3[i*M_SIZE+column];
    }
    GM1[id] = x;
}
//CUDA,SharedMemory使用版行列の積
__global__ void cudaMatrixMulShared(int *GM1, int *GM2, int *GM3){
    __shared__ int SM2[M_SIZE*SUB_SIZE], SM3[M_SIZE*SUB_SIZE];
    unsigned int tid = threadIdx.x;
    //ここらへんに転置する処理
    //GlobalMem -> SharedMem
    for(int i=SUB_SIZE; i < SUB_SIZE; i++){
        SM2[tid + M_SIZE * i] = GM2[tid + M_SIZE * i + blockIdx.y * M_SIZE * SUB_SIZE];
        SM3[tid + M_SIZE * i] = GM3[tid + M_SIZE * i + blockIdx.x * M_SIZE * SUB_SIZE];
    }
    __syncthreads();
    if(blockIdx.x == 0 && blockIdx.y == 0){
        printf("SM2[0]=%d, ",SM2[0]);
        printf("SM2[1]=%d, ",SM2[1]);
        printf("SM2[2]=%d\n",SM2[2]);
        printf("SM3[0]=%d, ",SM3[0]);
        printf("SM3[1]=%d, ",SM3[1]);
        printf("SM3[2]=%d\n",SM3[2]);
       
    }
    __syncthreads();
    for(int i=SUB_SIZE; i < SUB_SIZE; i++){
        for(int j=SUB_SIZE; j < SUB_SIZE; j++){
            //総和をとる
            //今はGMに直接足しこんでいるが,内部的にはレジスタに取り込んで
            //足して戻してを繰り返していると思われ,非効率である
            //SMにおいて総和するか,
            //GMにおいてSMに戻して総和にするかは後で考える
            GM1[M_SIZE*(blockIdx.y*SUB_SIZE + i) + blockIdx.x*SUB_SIZE + j] += SM2[i*SUB_SIZE + tid] * SM3[j*SUB_SIZE + tid];
        }
    }
}
//CPU版行列の積
void matrixMul(int *HM1, int *HM2, int *HM3){
    int tmpHM[M_SIZE*M_SIZE];
    matrixZeros(tmpHM);
    matrixTranspose(HM3,tmpHM);//転置
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE; j++){
            for(int k=0; k<M_SIZE; k++){
                HM1[i*M_SIZE + j] += HM2[i*M_SIZE + k] * tmpHM[j*M_SIZE + k];
            }
        }
    }
}
void matrixZeros(int *HM){
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        HM[i]=0;
    }
}
//転置行列
void matrixTranspose(int *iMat, int *oMat){
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE; j++){
            oMat[j*M_SIZE + i] = iMat[i*M_SIZE + j];
        }
    }
}
int matrixDiffCount(int *HM1, int *HM2){
    for(int i=0; i<M_SIZE*M_SIZE; i++){
        if(HM1[i] - HM2[i]){
            return 0;
        }
    }
    return 1;
}
void printHM(int *HM){
    for(int i=0; i<M_SIZE; i++){
        for(int j=0; j<M_SIZE;j++){
            printf("%d,",HM[i*M_SIZE + j]);
        }
        printf("\n");
    }
    printf("\n");
}
