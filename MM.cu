//正方行列の積を求め.cpuとgpuで速度比較する

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define M_SIZE 1024//matrix size

//プロトタイプ
void matrixMul(int *HM1, int *HM2, int *HM3);
void zerosM(int *HM);
void printHM(int *HM);
void fprintM(char *fileName,int *HM);
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3);

//メイン関数
int main(void){
  srand(123);
  //Host Memory に Matrix 確保 HM1 = HM2 * HM3
  int *HM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
  int *HM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
  int *HM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);

  //Host Memoryにデータ格納
  zerosM(HM1);
  for(int i=0; i<M_SIZE*M_SIZE; i++){
    HM2[i] = rand()%10;
    HM3[i] = rand()%10;
  }

  //Global Memory 確保
  int *GM1,*GM2,*GM3;
  hipMalloc((void **)&GM1, sizeof(int) * M_SIZE * M_SIZE);
  hipMalloc((void **)&GM2, sizeof(int) * M_SIZE * M_SIZE);
  hipMalloc((void **)&GM3, sizeof(int) * M_SIZE * M_SIZE);
  
  //GlobalMemoryにデータ格納
  hipMemcpy(GM1, HM1, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(GM2, HM2, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(GM3, HM3, sizeof(int) * M_SIZE * M_SIZE, hipMemcpyHostToDevice);

  //CPUでの計算
  clock_t startTime,stopTime;
  startTime = clock();
  matrixMul(HM1, HM2, HM3);
  stopTime = clock();
  
  //CUDAでの計算
  hipEvent_t cudaStartTime, cudaStopTime;
  float cudaTime;
  dim3 Dg(5, 5, 1), Db(4, 4, 2);
  hipEventCreate(&cudaStartTime);
  hipEventCreate(&cudaStopTime);
  hipEventRecord(cudaStartTime, 0);
  cudaMatrixMul <<< Dg, Db>>> (GM1, GM2, GM3);
  hipEventRecord(cudaStopTime, 0);
  hipEventSynchronize(cudaStopTime);
  hipEventElapsedTime(&cudaTime, cudaStartTime, cudaStopTime);

  //標準出力
  /*
  puts("M1");
  printHM(HM1);
  puts("M2");
  printHM(HM2);
  puts("M3");
  printHM(HM3);*/
  printf("CPU:Time = %f\n", (double)(stopTime - startTime)/CLOCKS_PER_SEC);
  printf("GPU:Time = %f\n", cudaTime);
  
 
  fprintM("cpuMatrixMul.txt",HM1);
  
  //Host Memory開放
  free(HM1);
  free(HM2);
  free(HM3);
  
  //Global Memory開放
  hipFree(GM1);
  hipFree(GM2);
  hipFree(GM3);
  return 0;
}

//CUDA版行列の積
__global__ void cudaMatrixMul(int *GM1, int *GM2, int *GM3){
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int row = id/M_SIZE;
  int column = id%M_SIZE;
  int x=0;
  for(int i=0; i<M_SIZE; i++){
    x += GM2[row*M_SIZE+i] * GM3[i*M_SIZE+column];
  }
  GM1[id] = x;
}

//CPU版行列の積
void matrixMul(int *HM1, int *HM2, int *HM3){
  for(int i=0; i<M_SIZE; i++){
    for(int j=0; j<M_SIZE; j++){
      for(int k=0; k<M_SIZE; k++){
	HM1[i*M_SIZE + j] += HM2[i*M_SIZE + k] * HM3[k*M_SIZE + j];
      }
    }
  }
}
void zerosM(int *HM){
  for(int i=0; i<M_SIZE*M_SIZE; i++){
    HM[i]=0;
  }
}
void printHM(int *HM){
  for(int i=0; i<M_SIZE; i++){
    for(int j=0; j<M_SIZE;j++){
      printf("%d,",HM[i*M_SIZE + j]);
    }
    printf("\n");
  }
  printf("\n");
}
void fprintM(char *fileName,int *HM){
  FILE *fp;
  fp = fopen(fileName, "w");
  for(int i=0; i<M_SIZE; i++){
    for(int j=0; j<M_SIZE; j++){
      fscanf(fp, "%d ", &HM);
    }
    puts("");
  }
  puts("");
}