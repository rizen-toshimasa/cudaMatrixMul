//����������Ѥ���.cpu��gpu��®����Ӥ���

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define M_SIZE 3//matrix size
void matrixMul(int *HM1, int *HM2, int *HM3){
  for(int i=0; i<M_SIZE; i++){
    for(int j=0; j<M_SIZE; j++){
      for(int k=0; k<M_SIZE; k++){
	HM1[i*M_SIZE + j] += HM2[i*M_SIZE + k] * HM3[k*M_SIZE + j];
      }
    }
  }
}
void zerosM(int *HM){
  for(int i=0; i<M_SIZE*M_SIZE; i++){
    HM[i]=0;
  }
}
void printHM(int *HM){
  for(int i=0; i<M_SIZE; i++){
    for(int j=0; j<M_SIZE;j++){
      printf("%d,",HM[i*M_SIZE + j]);
    }
    printf("\n");
  }
  printf("\n");
}/*
__global__ void matrixMulGPU(int *GM1, int GM2, int GM3){
  int id = blockDimxx * blockIdx.x + threadIdx.x;
 
  }*/
int main(void){
  srand(123);
  //Host Memory �� Matrix ���� HM1 = HM2 * HM3
  int *HM1 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
  int *HM2 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);
  int *HM3 = (int *)malloc(sizeof(int) * M_SIZE * M_SIZE);

  //Host Memory�˥ǡ�����Ǽ
  zerosM(HM1);
  for(int i=0; i<M_SIZE*M_SIZE; i++){
    HM2[i] = rand()%10;
    HM3[i] = rand()%10;
  }
  //Global Memory ����
  int *GM1,*GM2,*GM3;
  hipMalloc((void **)&GM1, sizeof(int) * M_SIZE * M_SIZE);
  hipMalloc((void **)&GM2, sizeof(int) * M_SIZE * M_SIZE);
  hipMalloc((void **)&GM3, sizeof(int) * M_SIZE * M_SIZE);

  //
  matrixMul(HM1, HM2, HM3);
  puts("matrix multiplication(M1 = M2 * M3)");
  puts("M1");
  printHM(HM1);
  puts("M2");
  printHM(HM2);
  puts("M3");
  printHM(HM3);

  free(HM1);
  free(HM2);
  free(HM3);
  return 0;
}
